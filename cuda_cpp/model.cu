#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <float.h>
#include <string>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "utility.cuh"
#include "parameterData.h"
#include "liggghtsData.h"
#include "compartment.cuh"

using namespace std;

#define TWOWAYCOUPLING false

// MACROS 
// Calling macros for error check and dump data to files to VaribleName.txt

#define DUMP(varName) dumpData(varName, #varName)
#define DUMP2D(varName) dump2DData(varName, #varName)
#define DUMP3D(varName) dump3DData(varName, #varName)

#define DUMPCSV(varName) dumpCSV(varName, #varName)
#define DUMP2DCSV(varName) dump2DCSV(varName, #varName)
#define DUMP3DCSV(varName) dump3DCSV(varName, #varName)

#define DUMPDIACSV(time, dia) dumpDiaCSV(time, dia, #dia)

#define DUMP2DCSV4MATLAB(varName) dump2DCSV4Matlab(varName, #varName)

// extern __shared__ double *d_sMeshXY, *d_ssMeshXY;


__global__ void initialization_kernel(double *d_vs, double *d_vss, size_t size2D, double fsVolCoeff, double ssVolCoeff, double fsVolBase, double ssVolBase, double *d_sAgg, 
                                      double *d_ssAgg, int *d_sAggregationCheck, int *d_ssAggregationCheck, double *d_sLow, double *d_ssLow, double *d_sHigh, double *d_ssHigh, 
                                      double *d_sMeshXY, double *d_ssMeshXY, int *d_sLoc, int *d_ssLoc, int *d_sInd, int *d_ssInd, double *d_sBreak, double *d_ssBreak, 
                                      int *d_sLocBreak, int *d_ssLocBreak, int *d_sCheckB, int*d_ssCheckB, int  *d_sIndB, int *d_ssIndB)
{
    int idx = threadIdx.x;
    int bix = blockIdx.x;
    int bdx = blockDim.x;

    // __shared__ double d_sMeshXY[256], d_ssMeshXY[256];

    d_sMeshXY[bdx * bix + idx] = d_vs[bix];
    d_ssMeshXY[bdx * bix + idx] = d_vss[bix];
    d_sAgg[bdx * bix + idx] = d_vs[idx] + d_vs[bix];
    d_ssAgg[bdx * bix + idx] = d_vss[idx] + d_vss[bix];
    d_sAggregationCheck[bdx * bix + idx] = d_sAgg[bdx * bix + idx] <= d_vs[bdx - 1] ? 1 : 0;
    d_ssAggregationCheck[bdx * bix + idx] = d_ssAgg[bdx * bix + idx] <= d_vss[bdx - 1] ? 1 : 0;
    d_sLow [bdx * bix + idx] = d_sMeshXY[bdx * bix + idx];
    d_ssLow[bdx * bix + idx] = d_ssMeshXY[bdx * bix + idx];
    __syncthreads();
    if (bix < bdx -1)
    {
        d_sHigh[bdx * bix + idx] = d_sMeshXY[bdx * (bix + 1) + idx];
        d_ssHigh[bdx * bix + idx] = d_sMeshXY[bdx * (bix + 1) + idx];
    }
    d_sHigh[bdx * (bdx -1) + idx] = 0.0;
    d_ssHigh[bdx * (bdx -1) + idx] = 0.0;
    d_sLoc[bdx * bix + idx] = floor(log(d_sAgg[bdx * bix + idx] / fsVolCoeff) / log(fsVolBase) + 1);
    d_ssLoc[bdx * bix + idx] = floor(log(d_ssAgg[bdx * bix + idx] / ssVolCoeff) / log(ssVolBase) + 1);
    d_sInd[bdx * bix + idx] = (idx <= bix) ? (bix + 1) : (idx + 1);
    d_ssInd[bdx * bix + idx] = (idx <= bix) ? (bix + 1) : (idx + 1);
    __syncthreads();
    double value = d_vs[idx] - d_vs[bix];
    double value1 = d_vss[idx] - d_vss[bix];
    d_sBreak[bdx * bix + idx] = value < 0.0 ? 0.0 : value;
    d_ssBreak[bdx * bix + idx] = value1 < 0.0 ? 0.0 : value1;
    d_sLocBreak[bdx * bix + idx] = (d_sBreak[bdx * idx + bix] == 0) ? 0 : (floor(log(d_sBreak[bdx * idx + bix] / fsVolCoeff) / log(fsVolBase) + 1));
    d_ssLocBreak[bdx * bix + idx] = (d_ssBreak[bdx * idx + bix] == 0) ? 0 : (floor(log(d_ssBreak[bdx * idx + bix] / ssVolCoeff) / log(ssVolBase) + 1));
    __syncthreads();
    d_sCheckB[bdx * bix + idx] = d_sLocBreak[bdx * bix + idx] >= 1 ? 1 : 0;
    d_ssCheckB[bdx * bix + idx] = d_ssLocBreak[bdx * bix + idx] >= 1 ? 1 : 0;
    d_sIndB[bdx * bix + idx] = d_sLocBreak[bdx * bix + idx];
    d_ssIndB[bdx * bix + idx] = d_ssLocBreak[bdx * bix + idx];
    if (d_sIndB[bdx * bix + idx] < 1)
        d_sIndB[bdx * bix + idx] = bdx + 1;
    if (d_ssIndB[bdx * bix + idx] < 1)
        d_ssIndB[bdx * bix + idx] = bdx + 1;
}



__global__ void launchCompartment(PreviousCompartmentIn *prevCompInData, CompartmentIn *compartmentIn, CompartmentDEMIn *compartmentDEMIn, double time, double timeStep, double initialTime, 
                                double *d_formationThroughAggregation, double *d_depletionThroughAggregation, double *d_formationThroughBreakage, double *d_depletionThroughBreakage,
                                double *d_fAllCompartments, double *d_flAllCompartments, double *d_fgAllCompartments, double *d_liquidAdditionRateAllCompartments, size_t size2D, size_t size3D, 
                                size_t size4D, double *d_fIn, double initPorosity) 
{
    int bix = blockIdx.x;
    int biy = blockIdx.y;
    int bdx = blockDim.x;
    int bdy = blockDim.y;

    int tix = threadIdx.x;
    int tiy = threadIdx.y;
    int dimx = gridDim.x;
    int dimy = gridDim.y;

    int idx = bix * bdx * bdy + tiy * bdx + tix;
    int ddx = bix * bdx + tix;

    if (tiy == 0)
    {
        compartmentIn->fAll[tix] = d_fAllCompartments[ddx];
        compartmentIn->fLiquid[tix] = d_flAllCompartments[ddx];
        compartmentIn->fGas[tix] = d_fgAllCompartments[ddx];
        compartmentIn->liquidAdditionRate = d_liquidAdditionRateAllCompartments[ddx];

        if (bix == 0)
        {
            prevCompInData->fAllComingIn[tix] = d_fIn[tix];
            double value = initPorosity * timeStep;
            // prevCompInData->fgComingIn[tix] = d_fIn[tix] * (compartmentIn->fAll)
        }
    }

    


    
}


int main(int argc, char *argv[])
{
    cout << "Code begins..." << endl;
    // Read passed arguments
    string startTimeStr;
    double startTime = 0.0;
    liggghtsData *lData = nullptr;
    parameterData *pData = nullptr;

    string coreVal;
    string diaVal;
    string pbmInFilePath;
    string timeVal;

    if (argc <5)
    {
        cout << "All values are not available as imput parameters " << endl;
        return 1;
    }

    pbmInFilePath = string(argv[1]);
    coreVal = string(argv[2]);
    diaVal = string(argv[3]);
    timeVal = string(argv[4]);

    pData = parameterData::getInstance();
    pData->readPBMInputFile(pbmInFilePath);

    int nCompartments = pData->nCompartments;
    CompartmentIn compartmentIn, *d_compartmentIn;
    PreviousCompartmentIn prevCompInData, *d_prevCompInData;
    CompartmentOut compartmentOut, *d_compartmentOut;
    CompartmentDEMIn compartmentDEMIn, *d_compartmentDEMIn;

    unsigned int nFirstSolidBins = pData->nFirstSolidBins;
    unsigned int nSecondSolidBins = pData->nSecondSolidBins;

    size_t size1D = nFirstSolidBins;
    size_t size2D = nFirstSolidBins * nSecondSolidBins;
    size_t size3D = nFirstSolidBins * nSecondSolidBins * nCompartments;
    size_t size4D = nFirstSolidBins * nFirstSolidBins * nSecondSolidBins * nSecondSolidBins;

    vector<double> h_vs(size1D, 0.0);
    vector<double> h_vss(size1D, 0.0);
    
    // Bin Initiation
    double fsVolCoeff = pData->fsVolCoeff;
    double fsVolBase = pData->fsVolBase;
    for (size_t i = 0; i < nFirstSolidBins; i++)
        h_vs[i] = fsVolCoeff * pow(fsVolBase, i); // m^3

    double ssVolCoeff = pData->ssVolCoeff;
    double ssVolBase = pData->ssVolBase;
    for (size_t i = 0; i < nSecondSolidBins; i++) 
        h_vss[i] = ssVolCoeff * pow(ssVolBase, i); // m^3

    arrayOfDouble2D diameter1 = getArrayOfDouble2D(nFirstSolidBins, nSecondSolidBins);
    for (size_t s = 0; s < nFirstSolidBins; s++)
        for (size_t ss = 0; ss < nSecondSolidBins; ss++)
            diameter1[s][ss] = cbrt((6/M_PI) * (h_vs[s] + h_vss[ss]));

    vector<double> diameter = linearize2DVector(diameter1);
    
    vector<double> particleIn;
    particleIn.push_back(726657587.0);
    particleIn.push_back(286654401.0);
    particleIn.push_back(118218011.0);
    particleIn.push_back(50319795.0);
    particleIn.push_back(20954036.0);
    particleIn.push_back(7345998.0);
    particleIn.push_back(1500147.0);
    particleIn.push_back(76518.0);
    particleIn.push_back(149.0);
    
    vector<double> h_fIn(size2D, 0.0);
    for (size_t i = 0; i < size2D; i++)
        h_fIn[i * size1D + i] = particleIn[i];
    
    // allocation of memory for the matrices that will be copied onto the device from the host
    double *d_vs = device_alloc_double_vector(size1D);
    double *d_vss = device_alloc_double_vector(size1D);
    
    double *d_sMeshXY = device_alloc_double_vector(size2D);
    double *d_ssMeshXY = device_alloc_double_vector(size2D);

    double *d_sAgg = device_alloc_double_vector(size2D);
    double *d_ssAgg = device_alloc_double_vector(size2D);

    int *d_sAggregationCheck = device_alloc_integer_vector(size2D);
    int *d_ssAggregationCheck = device_alloc_integer_vector(size2D);

    double *d_sLow = device_alloc_double_vector(size2D);
    double *d_ssLow = device_alloc_double_vector(size2D);

    double *d_sHigh = device_alloc_double_vector(size2D);
    double *d_ssHigh = device_alloc_double_vector(size2D);

    int *d_sLoc = device_alloc_integer_vector(size2D);
    int *d_ssLoc = device_alloc_integer_vector(size2D);

    int *d_sInd = device_alloc_integer_vector(size2D);
    int *d_ssInd = device_alloc_integer_vector(size2D);

    double *d_sBreak = device_alloc_double_vector(size2D);
    double *d_ssBreak = device_alloc_double_vector(size2D);

    int *d_sLocBreak = device_alloc_integer_vector(size2D);
    int *d_ssLocBreak = device_alloc_integer_vector(size2D);

    int *d_sCheckB = device_alloc_integer_vector(size2D);
    int *d_ssCheckB = device_alloc_integer_vector(size2D);

    int *d_sIndB = device_alloc_integer_vector(size2D);
    int *d_ssIndB = device_alloc_integer_vector(size2D);

    // defining vectors for data required for compartment calculations
    vector<double> h_sMeshXY(size2D, 0.0);
    vector<double> h_ssMeshXY(size2D, 0.0);

    vector<int> h_sAggregationCheck(size2D, 0);
    vector<int> h_ssAggregationCheck(size2D, 0);

    vector<double> h_sLow(size2D, 0.0);
    vector<double> h_ssLow(size2D, 0.0);

    vector<double> h_sHigh(size2D, 0.0);
    vector<double> h_ssHigh(size2D, 0.0);

    vector<int> h_sInd(size2D, 0);
    vector<int> h_ssInd(size2D, 0);

    vector<int> h_sCheckB(size2D, 0);
    vector<int> h_ssCheckB(size2D, 0);

    vector<int> h_sIndB(size2D, 0.0);
    vector<int> h_ssIndB(size2D, 0.0);

    copy_double_vector_fromHtoD(d_vs, h_vs.data(), size1D);
    copy_double_vector_fromHtoD(d_vss, h_vss.data(), size1D);

    int nBlocks = nFirstSolidBins;
    int nThreads = nSecondSolidBins;

    initialization_kernel<<<nBlocks,nThreads>>>(d_vs, d_vss, size2D, fsVolCoeff, ssVolCoeff, fsVolBase, ssVolBase, d_sAgg,d_ssAgg, d_sAggregationCheck, d_ssAggregationCheck, 
                                    d_sLow, d_ssLow, d_sHigh, d_ssHigh, d_sMeshXY, d_ssMeshXY, d_sLoc, d_ssLoc, d_sInd, d_ssInd, d_sBreak, d_ssBreak, d_sLocBreak, d_ssLocBreak,
                                    d_sCheckB, d_ssCheckB, d_sIndB, d_ssIndB);
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch initialization kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    cout << "Initialization complete" << endl;

    // copy back data required for the compartment calculations
     
    copy_double_vector_fromDtoH(h_sMeshXY.data(), d_sMeshXY, size2D);
    copy_double_vector_fromDtoH(h_ssMeshXY.data(), d_ssMeshXY, size2D);

    copy_integer_vector_fromDtoH(h_sAggregationCheck.data(), d_sAggregationCheck, size2D);
    copy_integer_vector_fromDtoH(h_ssAggregationCheck.data(), d_ssAggregationCheck, size2D);

    copy_double_vector_fromDtoH(h_sLow.data(), d_sLow, size2D);
    copy_double_vector_fromDtoH(h_ssLow.data(), d_ssLow, size2D);

    copy_double_vector_fromDtoH(h_sHigh.data(), d_sHigh, size2D);
    copy_double_vector_fromDtoH(h_ssHigh.data(), d_ssHigh, size2D);

    copy_integer_vector_fromDtoH(h_sInd.data(), d_sInd, size2D);
    copy_integer_vector_fromDtoH(h_ssInd.data(), d_ssInd, size2D);

    copy_integer_vector_fromDtoH(h_sCheckB.data(), d_sCheckB, size2D);
    copy_integer_vector_fromDtoH(h_ssCheckB.data(), d_ssCheckB, size2D);

    copy_integer_vector_fromDtoH(h_sIndB.data(), d_sIndB, size2D);
    copy_integer_vector_fromDtoH(h_ssIndB.data(), d_ssIndB, size2D);

    hipDeviceSynchronize();

    vector<double> h_fAllCompartments(size3D, 0.0);
    vector<double> h_flAllCompartments(size3D, 0.0);
    vector<double> h_fgAllCompartments(size3D, 0.0);

    vector<double> h_dfdtAllCompartments(size3D, 0.0);
    vector<double> h_dfldtAllCompartments(size3D, 0.0);
    vector<double> h_dfgdtAllCompartments(size3D, 0.0);
    
    vector<double> h_externalVolumeBinsAllCompartments(size3D, 0.0);
    vector<double> h_internalVolumeBinsAllCompartments(size3D, 0.0);
    vector<double> h_liquidBinsAllCompartments(size3D, 0.0);
    vector<double> h_gasBinsAllCompartments(size3D, 0.0);
    vector<double> h_totalVolumeBinsAllCompartments(size3D, 0.0);
    
    vector<double> h_internalLiquidAllCompartments(size3D, 0.0);
    vector<double> h_externalLiquidAllCompartments(size3D, 0.0);
    
    vector<double> h_internalVolumeBins(size2D, 0.0);
    vector<double> h_externalVolumeBins(size2D, 0.0);

    lData = liggghtsData::getInstance();
    lData->readLiggghtsDataFiles(coreVal, diaVal);

    vector<double> DEMDiameter = lData->getDEMParticleDiameters();
    if ((DEMDiameter).size() == 0)
    {
        cout << "Diameter data is missing in LIGGGHTS output file" << endl;
        cout << "Input parameters for DEM core and diameter aren't matching with LIGGGHTS output file" << endl;
        return 1;
    }

    vector<double> DEMImpactData = lData->getFinalDEMImpactData();
    if ((DEMImpactData).size() == 0)
    {
        cout << "Impact data is missing in LIGGGHTS output file" << endl;
        cout << "Input parameters for DEM core and diameter aren't matching with LIGGGHTS output file" << endl;
        return 1;
    }

    arrayOfDouble2D DEMCollisionData = lData->getFinalDEMCollisionData();
    if (DEMCollisionData.size() == 0)
    {
        cout << "Collision data is missing in LIGGGHTS output file" << endl;
        cout << "Input parameters for DEM core and diameter aren't matching with LIGGGHTS output file" << endl;
        return 1;
    }
    vector<double> velocity = lData->getFinalDEMImpactVelocity();
    if (velocity.size() == 0)
    {
        cout << "Velocity is missing in LIGGGHTS output file" << endl;
        cout << "Input parameters for DEM core and diameter aren't matching with LIGGGHTS output file" << endl;
	    return 1;
    }

    DUMP2D(DEMCollisionData);
    DUMP(DEMDiameter);
    DUMP(DEMImpactData);
    DUMP(velocity);

    //Initialize DEM data for compartment
    compartmentDEMIn.DEMDiameter = DEMDiameter.data();
    compartmentDEMIn.DEMCollisionData = linearize2DVector(DEMCollisionData).data();
    compartmentDEMIn.DEMImpactData = DEMImpactData.data();

    vector<double> liquidAdditionRateAllCompartments(nCompartments, 0.0);
    double liqSolidRatio = pData->liqSolidRatio;
    double throughput = pData->throughput;
    double liqDensity = pData->liqDensity;
    double liquidAddRate = (liqSolidRatio * throughput) / (liqDensity * 3600);
    liquidAdditionRateAllCompartments[0] = liquidAddRate;
    
    vector<double> h_fAllCompartmentsOverTime(size4D, 0.0);
    vector<double> h_externalVolumeBinsAllCompartmentsOverTime(size4D, 0.0);
    vector<double> h_internalVolumeBinsAllCompartmentsOverTime(size4D, 0.0);
    vector<double> h_liquidBinsAllCompartmentsOverTime(size4D, 0.0);
    vector<double> h_gasBinsAllCompartmentsOverTime(size4D, 0.0);

    double granulatorLength = pData->granulatorLength;
    double partticleResTime = pData->partticleResTime;
    double particleAveVelo = granulatorLength /  partticleResTime;
    vector<double> particleAverageVelocity(nCompartments, particleAveVelo);


    //Initialize input data for compartment

    compartmentIn.vs = h_vs.data();
    compartmentIn.vss = h_vss.data();

    compartmentIn.diameter = diameter.data();

    compartmentIn.sMeshXY = h_sMeshXY.data();
    compartmentIn.ssMeshXY = h_ssMeshXY.data();

    compartmentIn.sAggregationCheck = h_sAggregationCheck.data();
    compartmentIn.ssAggregationCheck = h_ssAggregationCheck.data();

    compartmentIn.sLow = h_sLow.data();
    compartmentIn.sHigh = h_sHigh.data();

    compartmentIn.ssLow = h_ssLow.data();
    compartmentIn.ssHigh = h_ssHigh.data();

    compartmentIn.sInd = h_sInd.data();
    compartmentIn.ssInd = h_ssInd.data();

    compartmentIn.sCheckB = h_sCheckB.data();
    compartmentIn.ssCheckB = h_ssCheckB.data();

    compartmentIn.sIndB = h_sIndB.data();
    compartmentIn.ssIndB = h_ssIndB.data();

    vector<int> sieveGrid;
    sieveGrid.push_back(38);
    sieveGrid.push_back(63);
    sieveGrid.push_back(90);
    sieveGrid.push_back(125);
    sieveGrid.push_back(250);
    sieveGrid.push_back(355);
    sieveGrid.push_back(500);
    sieveGrid.push_back(710);
    sieveGrid.push_back(850);
    sieveGrid.push_back(1000);
    sieveGrid.push_back(1400);
    sieveGrid.push_back(2000);
    sieveGrid.push_back(2380);
    sieveGrid.push_back(4000);
    size_t nSieveGrid = sieveGrid.size();

    vector<double> d10OverTime(size2D, 0.0);
    vector<double> d50OverTime(size2D, 0.0);
    vector<double> d90OverTime(size2D, 0.0);

    double time = stod(timeVal); // initial time to start PBM
    double timeStep = 0.5; //1.0e-1;
    vector<double> Time;

    double lastTime = time;
    int timeIdxCount = 0;
    int lastTimeIdxCount = 0;

    double premixTime = pData->premixTime;
    double liqAddTime = pData->liqAddTime;
    double postMixTime = pData->postMixTime;
    double finalTime = premixTime + liqAddTime + postMixTime + stod(timeVal);
    double initPorosity = pData->initPorosity;
    
    vector<double> formationThroughAggregationOverTime;
    vector<double> depletionThroughAggregationOverTime;
    vector<double> formationThroughBreakageOverTime;
    vector<double> depletionThroughBreakageOverTime;
    cout << "time" << endl;

    dim3 compKernel_nblocks, compKernel_nthreads;
    compKernel_nblocks = dim3(nCompartments,1,1);
    compKernel_nthreads = dim3(size2D, size2D,1);

    vector<double> temp(size2D, 0);
    prevCompInData.fAllPreviousCompartment = temp.data();
    prevCompInData.flPreviousCompartment = temp.data();
    prevCompInData.fgPreviousCompartment = temp.data();
    prevCompInData.fAllComingIn = temp.data();
    prevCompInData.fgComingIn = temp.data();

    // int compKernel_nblocks = nCompartments;
    // int compKernel_nthreads = size2D * size2D;

    // allocating memory for structures used for compartment calculations

    err = hipMalloc(&d_compartmentIn, sizeof(CompartmentIn));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to  hipMalloc : CompartmentIn (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc(&d_prevCompInData, sizeof(PreviousCompartmentIn));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to  hipMalloc : prevCompInData (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc(&d_compartmentDEMIn, sizeof(CompartmentDEMIn));
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to  hipMalloc : compartmentDEMIn (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // copying data to the allocated GPU

    hipMemcpy(d_compartmentIn, &compartmentIn, sizeof(CompartmentIn), hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to hipMemcpy : CompartmentIn (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_prevCompInData, &prevCompInData, sizeof(PreviousCompartmentIn), hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to hipMemcpy : PreviousCompartmentIn (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_compartmentDEMIn, &compartmentDEMIn, sizeof(CompartmentDEMIn), hipMemcpyHostToDevice);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to hipMemcpy : CompartmentDEMIn (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    vector<double> h_formationThroughAggregation(nCompartments, 0.0);
    vector<double> h_depletionThroughAggregation(nCompartments, 0.0);
    vector<double> h_formationThroughBreakage(nCompartments, 0.0);
    vector<double> h_depletionThroughBreakage(nCompartments, 0.0);

    double *d_formationThroughAggregation = device_alloc_double_vector(nCompartments);
    double *d_depletionThroughAggregation = device_alloc_double_vector(nCompartments);
    double *d_formationThroughBreakage = device_alloc_double_vector(nCompartments);
    double *d_depletionThroughBreakage = device_alloc_double_vector(nCompartments);

    double *d_fAllCompartments = device_alloc_double_vector(size3D);
    double *d_flAllCompartments = device_alloc_double_vector(size3D);
    double *d_fgAllCompartments = device_alloc_double_vector(size3D);
    double *d_liquidAdditionRateAllCompartments = device_alloc_double_vector(nCompartments);

    double *d_fIn = device_alloc_double_vector(size2D);

    copy_double_vector_fromHtoD(d_liquidAdditionRateAllCompartments, liquidAdditionRateAllCompartments.data(), nCompartments);
    copy_double_vector_fromHtoD(d_fIn, h_fIn.data(), size2D);


    while (time <= finalTime)
    {
        copy_double_vector_fromHtoD(d_fAllCompartments, h_fAllCompartments.data(), size3D);
        copy_double_vector_fromHtoD(d_flAllCompartments, h_flAllCompartments.data(), size3D);
        copy_double_vector_fromHtoD(d_fgAllCompartments, h_fgAllCompartments.data(), size3D);

        launchCompartment<<<compKernel_nblocks,compKernel_nthreads>>>(d_prevCompInData, d_compartmentIn, d_compartmentDEMIn, time, timeStep, stod(timeVal),
                            d_formationThroughAggregation, d_depletionThroughAggregation, d_formationThroughBreakage, d_depletionThroughBreakage, d_fAllCompartments, 
                            d_flAllCompartments, d_fgAllCompartments, d_liquidAdditionRateAllCompartments, size2D, size3D, size4D, d_fIn, initPorosity);

    }

    hipFree(d_vs);
    hipFree(d_vss);
    // hipFree(d_sMeshXY);
    // hipFree(d_ssMeshXY);
}