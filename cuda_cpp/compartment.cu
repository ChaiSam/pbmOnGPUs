#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>
#include <cmath>

#include "parameterData.h"
#include "utility.cuh"
#include "compartment.cuh"

using namespace std;

__global__ void performAggCalculations(PreviousCompartmentIn *d_prevCompIn, CompartmentIn *d_compartmentIn, CompartmentDEMIn *d_compartmentDEMIn, 
                                        CompartmentOut *d_compartmentOut, CompartmentVar *d_compVar, AggregationCompVar *d_aggCompVar, 
                                        double time, double timeStep, double initialTime, double demTimeStep)
{
    int bix = blockIdx.x;
    int biy = blockIdx.y;
    int bdx = blockDim.x;
    int bdy = blockDim.y;

    int tix = threadIdx.x;
    int tiy = threadIdx.y;
    int dimx = gridDim.x;
    int dimy = gridDim.y;

    int idx4 = tix * bdx + tix;

    double criticalExternalLiquid = 0.2;
    bool flag1 = (d_compartmentIn->fAll[tix] >= 0.0) && (d_compartmentIn->fAll[tiy] >= 0.0);
    bool flag2 = ((d_compVar->externalLiquid[tix] + d_compVar->externalLiquid[tiy]) / (d_compartmentIn->fAll[tiy] * d_compartmentIn->vs[tiy % 16] + d_compartmentIn->fAll[tix] * d_compartmentIn->vss[tix % 16]));
    bool flag3 = (d_compartmentDEMIn->velocityCol[tiy % 16] < d_compartmentDEMIn->uCriticalCol);
    if (flag1 && flag2 && flag3)
    {
        d_compartmentDEMIn->colEfficiency[idx4] = d_compartmentDEMIn->colProbability[tix % 16];
    }
    else
        d_compartmentDEMIn->colEfficiency[idx4] = 0.0;
    d_compartmentDEMIn->colFrequency[idx4] = (d_compartmentDEMIn->DEMCollisionData[tix] * timeStep) / demTimeStep;

    d_compartmentOut->aggregationKernel[idx4] = d_aggCompVar->aggKernelConst * d_compartmentDEMIn->colFrequency[idx4] * d_compartmentDEMIn->colEfficiency[idx4];
    printf("Value of kernel at %d and %d is %f \n", tix, tiy, d_compartmentOut->aggregationKernel[idx4]);
}



// ============ Constructors for the Classes =================


CompartmentVar :: CompartmentVar(unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        internalLiquid = alloc_double_vector(nX2);
        externalLiquid = alloc_double_vector(nX2);
        externalLiquidContent = alloc_double_vector(nX2);
        volumeBins = alloc_double_vector(nX2);
        aggregationRate = alloc_double_vector(nX4);
        breakageRate = alloc_double_vector(nX4);
        particleMovement = alloc_double_vector(nX2);
        liquidMovement = alloc_double_vector(nX2);
        gasMovement = alloc_double_vector(nX2);
        liquidBins = alloc_double_vector(nX2);
        gasBins = alloc_double_vector(nX2);
    }

    else if (check == 1)
    {
        internalLiquid = device_alloc_double_vector(nX2);
        externalLiquid = device_alloc_double_vector(nX2);
        externalLiquidContent = device_alloc_double_vector(nX2);
        volumeBins = device_alloc_double_vector(nX2);
        aggregationRate = device_alloc_double_vector(nX4);
        breakageRate = device_alloc_double_vector(nX4);
        particleMovement = device_alloc_double_vector(nX2);
        liquidMovement = device_alloc_double_vector(nX2);
        gasMovement = device_alloc_double_vector(nX2);
        liquidBins = device_alloc_double_vector(nX2);
        gasBins = device_alloc_double_vector(nX2);
    }

    else
        printf("\n Wrong Value of check passed in Compartment Var call \n");
}

CompartmentIn :: CompartmentIn (unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        fAll = alloc_double_vector(nX2);
        fLiquid = alloc_double_vector(nX2);
        fGas = alloc_double_vector(nX2);
        liquidAdditionRate = 0.0;
        vs = alloc_double_vector(nX2);
        vss = alloc_double_vector(nX2);
        sMeshXY = alloc_double_vector(nX2);
        ssMeshXY = alloc_double_vector(nX2);
        sAggregationCheck = alloc_integer_vector(nX2);
        ssAggregationCheck = alloc_integer_vector(nX2);;
        sInd = alloc_integer_vector(nX2);;
        ssInd = alloc_integer_vector(nX2);;
        sIndB = alloc_integer_vector(nX2);;
        ssIndB = alloc_integer_vector(nX2);;
        sLow = alloc_double_vector(nX2);
        sHigh = alloc_double_vector(nX2);
        ssLow = alloc_double_vector(nX2);
        ssHigh = alloc_double_vector(nX2);
        sCheckB = alloc_integer_vector(nX2);;
        ssCheckB = alloc_integer_vector(nX2);;
        diameter = alloc_double_vector(nX2);
    }

    else if (check == 1)
    {
        liquidAdditionRate = 0.0;
        fAll = device_alloc_double_vector(nX2);
        fLiquid = device_alloc_double_vector(nX2);
        fGas = device_alloc_double_vector(nX2);
        vs = device_alloc_double_vector(nX2);
        vss = device_alloc_double_vector(nX2);
        sMeshXY = device_alloc_double_vector(nX2);
        ssMeshXY = device_alloc_double_vector(nX2);
        sAggregationCheck = device_alloc_integer_vector(nX2);
        ssAggregationCheck = device_alloc_integer_vector(nX2);;
        sInd = device_alloc_integer_vector(nX2);;
        ssInd = device_alloc_integer_vector(nX2);;
        sIndB = device_alloc_integer_vector(nX2);;
        ssIndB = device_alloc_integer_vector(nX2);;
        sLow = device_alloc_double_vector(nX2);
        sHigh = device_alloc_double_vector(nX2);
        ssLow = device_alloc_double_vector(nX2);
        ssHigh = device_alloc_double_vector(nX2);
        sCheckB = device_alloc_integer_vector(nX2);;
        ssCheckB = device_alloc_integer_vector(nX2);;
        diameter = device_alloc_double_vector(nX2);
    }

    else
        printf("\n Wrong Value of check passed in CompartmentIn  call \n");
}

PreviousCompartmentIn :: PreviousCompartmentIn(unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        fAllPreviousCompartment = alloc_double_vector(nX2);
        flPreviousCompartment = alloc_double_vector(nX2);
        fgPreviousCompartment = alloc_double_vector(nX2);
        fAllComingIn = alloc_double_vector(nX2);
        fgComingIn = alloc_double_vector(nX2);
    }

    else if (check == 1)
    {
        fAllPreviousCompartment = device_alloc_double_vector(nX2);
        flPreviousCompartment = device_alloc_double_vector(nX2);
        fgPreviousCompartment = device_alloc_double_vector(nX2);
        fAllComingIn = device_alloc_double_vector(nX2);
        fgComingIn = device_alloc_double_vector(nX2);
    }
    else
        printf("\n Wrong Value of check passed in PreviousCompartmentIn  call \n");

}

CompartmentDEMIn :: CompartmentDEMIn(unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        DEMDiameter = alloc_double_vector(sqrt(nX2));
        DEMCollisionData = alloc_double_vector(nX2);
        DEMImpactData = alloc_double_vector(sqrt(nX2));
        colProbability = alloc_double_vector(sqrt(nX2));
        brProbability = alloc_double_vector(sqrt(nX2));
        colEfficiency = alloc_double_vector(nX4);
        colFrequency = alloc_double_vector(nX4);
        velocityCol = alloc_double_vector(sqrt(nX2));
        uCriticalCol = 0.0;
    }

    else if (check == 1)
    {
        DEMDiameter = device_alloc_double_vector(sqrt(nX2));
        DEMCollisionData = device_alloc_double_vector(nX2);
        DEMImpactData = device_alloc_double_vector(sqrt(nX2));
        colProbability = device_alloc_double_vector(sqrt(nX2));
        brProbability = device_alloc_double_vector(sqrt(nX2));
        colEfficiency = device_alloc_double_vector(nX4);
        colFrequency = device_alloc_double_vector(nX4);
        velocityCol = device_alloc_double_vector(sqrt(nX2));
        uCriticalCol = 0.0;
    }

    else 
        printf("\n Wrong Value of check passed in CompartmentDEMIn call \n");

}

CompartmentOut :: CompartmentOut(unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        dfAlldt = alloc_double_vector(nX2);
        dfLiquiddt = alloc_double_vector(nX2);
        dfGasdt = alloc_double_vector(nX2);
        liquidBins = alloc_double_vector(nX2);
        gasBins = alloc_double_vector(nX2);
        internalVolumeBins = alloc_double_vector(nX2);
        externalVolumeBins = alloc_double_vector(nX2);
        aggregationKernel = alloc_double_vector(nX4);
        breakageKernel = alloc_double_vector(nX4);
        collisionFrequency = alloc_double_vector(nX4);
        formationThroughAggregation = 0.0;
        depletionThroughAggregation = 0.0;
        formationThroughBreakage = 0.0;
        depletionThroughBreakage = 0.0;
    }

    else if (check == 1)
    {
        dfAlldt = device_alloc_double_vector(nX2);
        dfLiquiddt = device_alloc_double_vector(nX2);
        dfGasdt = device_alloc_double_vector(nX2);
        liquidBins = device_alloc_double_vector(nX2);
        gasBins = device_alloc_double_vector(nX2);
        internalVolumeBins = device_alloc_double_vector(nX2);
        externalVolumeBins = device_alloc_double_vector(nX2);
        aggregationKernel = device_alloc_double_vector(nX4);
        breakageKernel = device_alloc_double_vector(nX4);
        collisionFrequency = device_alloc_double_vector(nX4);
        formationThroughAggregation = 0.0;
        depletionThroughAggregation = 0.0;
        formationThroughBreakage = 0.0;
        depletionThroughBreakage = 0.0;
    }

    else
        printf("\n Wrong Value of check passed in CompartmentOut call \n");
}

BreakageCompVar :: BreakageCompVar(unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        birthThroughBreakage1 = alloc_double_vector(nX2);
        birthThroughBreakage2 = alloc_double_vector(nX2);
        firstSolidBirthThroughBreakage = alloc_double_vector(nX2);
        secondSolidBirthThroughBreakage = alloc_double_vector(nX2);
        liquidBirthThroughBreakage1 = alloc_double_vector(nX2);
        gasBirthThroughBreakage1 = alloc_double_vector(nX2);
        liquidBirthThroughBreakage2 = alloc_double_vector(nX2);
        gasBirthThroughBreakage2 = alloc_double_vector(nX2);
        firstSolidVolumeThroughBreakage = alloc_double_vector(nX2);
        secondSolidVolumeThroughBreakage = alloc_double_vector(nX2);
        fractionBreakage00 = alloc_double_vector(nX2);
        fractionBreakage01 = alloc_double_vector(nX2);
        fractionBreakage10 = alloc_double_vector(nX2);
        fractionBreakage11 = alloc_double_vector(nX2);
        formationThroughBreakageCA = alloc_double_vector(nX2);
        formationOfLiquidThroughBreakageCA = alloc_double_vector(nX2);
        formationOfGasThroughBreakageCA = alloc_double_vector(nX2);
        transferThroughLiquidAddition = alloc_double_vector(nX2);
        transferThroughConsolidation = alloc_double_vector(nX2);
    }

    else if (check == 1)
    {
        birthThroughBreakage1 = device_alloc_double_vector(nX2);
        birthThroughBreakage2 = device_alloc_double_vector(nX2);
        firstSolidBirthThroughBreakage = device_alloc_double_vector(nX2);
        secondSolidBirthThroughBreakage = device_alloc_double_vector(nX2);
        liquidBirthThroughBreakage1 = device_alloc_double_vector(nX2);
        gasBirthThroughBreakage1 = device_alloc_double_vector(nX2);
        liquidBirthThroughBreakage2 = device_alloc_double_vector(nX2);
        gasBirthThroughBreakage2 = device_alloc_double_vector(nX2);
        firstSolidVolumeThroughBreakage = device_alloc_double_vector(nX2);
        secondSolidVolumeThroughBreakage = device_alloc_double_vector(nX2);
        fractionBreakage00 = device_alloc_double_vector(nX2);
        fractionBreakage01 = device_alloc_double_vector(nX2);
        fractionBreakage10 = device_alloc_double_vector(nX2);
        fractionBreakage11 = device_alloc_double_vector(nX2);
        formationThroughBreakageCA = device_alloc_double_vector(nX2);
        formationOfLiquidThroughBreakageCA = device_alloc_double_vector(nX2);
        formationOfGasThroughBreakageCA = device_alloc_double_vector(nX2);
        transferThroughLiquidAddition = device_alloc_double_vector(nX2);
        transferThroughConsolidation = device_alloc_double_vector(nX2);
    }

    else
        printf("\n Wrong Value of check passed in BreakageCompVar call \n");
}

AggregationCompVar :: AggregationCompVar(unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        aggKernelConst = 0.0;
        depletionOfGasThroughAggregation = alloc_double_vector(nX2);
        depletionOfLiquidThroughAggregation = alloc_double_vector(nX2);
        birthThroughAggregation = alloc_double_vector(nX2);
        firstSolidBirthThroughAggregation = alloc_double_vector(nX2);
        secondSolidBirthThroughAggregation = alloc_double_vector(nX2);
        liquidBirthThroughAggregation = alloc_double_vector(nX2);
        gasBirthThroughAggregation = alloc_double_vector(nX2);
        firstSolidVolumeThroughAggregation = alloc_double_vector(nX2);
        secondSolidVolumeThroughAggregation = alloc_double_vector(nX2);
        birthAggLowLow = alloc_double_vector(nX2);
        birthAggHighHigh = alloc_double_vector(nX2);
        birthAggLowHigh = alloc_double_vector(nX2);
        birthAggHighLow = alloc_double_vector(nX2);
        birthAggLowLowLiq = alloc_double_vector(nX2);
        birthAggHighHighLiq = alloc_double_vector(nX2);
        birthAggLowHighLiq = alloc_double_vector(nX2);
        birthAggHighLowLiq = alloc_double_vector(nX2);
        birthAggLowLowGas = alloc_double_vector(nX2);
        birthAggHighHighGas = alloc_double_vector(nX2);
        birthAggLowHighGas = alloc_double_vector(nX2);
        birthAggHighLowGas = alloc_double_vector(nX2);
        formationThroughAggregationCA = alloc_double_vector(nX2);
        formationOfLiquidThroughAggregationCA = alloc_double_vector(nX2);
        formationOfGasThroughAggregationCA = alloc_double_vector(nX2);
        depletionThroughAggregation = alloc_double_vector(nX2);
        depletionThroughBreakage = alloc_double_vector(nX2);
        depletionOfGasThroughBreakage = alloc_double_vector(nX2);
        depletionOfLiquidthroughBreakage = alloc_double_vector(nX2);
    }

    else if (check == 1)
    {
        aggKernelConst = 0.0;
        depletionOfGasThroughAggregation = device_alloc_double_vector(nX2);
        depletionOfLiquidThroughAggregation = device_alloc_double_vector(nX2);
        birthThroughAggregation = device_alloc_double_vector(nX2);
        firstSolidBirthThroughAggregation = device_alloc_double_vector(nX2);
        secondSolidBirthThroughAggregation = device_alloc_double_vector(nX2);
        liquidBirthThroughAggregation = device_alloc_double_vector(nX2);
        gasBirthThroughAggregation = device_alloc_double_vector(nX2);
        firstSolidVolumeThroughAggregation = device_alloc_double_vector(nX2);
        secondSolidVolumeThroughAggregation = device_alloc_double_vector(nX2);
        birthAggLowLow = device_alloc_double_vector(nX2);
        birthAggHighHigh = device_alloc_double_vector(nX2);
        birthAggLowHigh = device_alloc_double_vector(nX2);
        birthAggHighLow = device_alloc_double_vector(nX2);
        birthAggLowLowLiq = device_alloc_double_vector(nX2);
        birthAggHighHighLiq = device_alloc_double_vector(nX2);
        birthAggLowHighLiq = device_alloc_double_vector(nX2);
        birthAggHighLowLiq = device_alloc_double_vector(nX2);
        birthAggLowLowGas = device_alloc_double_vector(nX2);
        birthAggHighHighGas = device_alloc_double_vector(nX2);
        birthAggLowHighGas = device_alloc_double_vector(nX2);
        birthAggHighLowGas = device_alloc_double_vector(nX2);
        formationThroughAggregationCA = device_alloc_double_vector(nX2);
        formationOfLiquidThroughAggregationCA = device_alloc_double_vector(nX2);
        formationOfGasThroughAggregationCA = device_alloc_double_vector(nX2);
        depletionThroughAggregation = device_alloc_double_vector(nX2);
        depletionThroughBreakage = device_alloc_double_vector(nX2);
        depletionOfGasThroughBreakage = device_alloc_double_vector(nX2);
        depletionOfLiquidthroughBreakage = device_alloc_double_vector(nX2);
    }
    else
        printf("\n Wrong Value of check passed in BreakageCompVar call \n");
}