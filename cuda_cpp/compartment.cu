#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>
#include <cmath>

#include "parameterData.h"
#include "utility.cuh"
#include "compartment.cuh"

using namespace std;

__global__ void performAggCalculations(PreviousCompartmentIn *d_prevCompIn, CompartmentIn *d_compartmentIn, CompartmentDEMIn *d_compartmentDEMIn, 
                                        CompartmentOut *d_compartmentOut, CompartmentVar *d_compVar, AggregationCompVar *d_aggCompVar, 
                                        double time, double timeStep, double initialTime, double demTimeStep, int blx, int tlx, int mbdx, int nFirstSolidBins, int nSecondSolidBins)
{
    int bix = blockIdx.x;
    int bdx = blockDim.x;

    int tix = threadIdx.x;
    int dimx = gridDim.x;

    int idx4 = blx * mbdx * bdx + tlx * bdx + tix;
    int idx3 = blx * mbdx + tlx;
    int idx3s = blx * bdx + tix;

    double criticalExternalLiquid = 0.2;
    bool flag1 = (d_compartmentIn->fAll[idx3] >= 0.0) && (d_compartmentIn->fAll[idx3] >= 0.0);
    bool flag2 = ((d_compVar->externalLiquid[idx3] + d_compVar->externalLiquid[idx3]) / (d_compartmentIn->fAll[idx3] * d_compartmentIn->vs[idx3 % nFirstSolidBins] + d_compartmentIn->fAll[idx3] * d_compartmentIn->vss[idx3 % 16]));
    bool flag3 = (d_compartmentDEMIn->velocityCol[idx3 % 16] < d_compartmentDEMIn->uCriticalCol);
    if (flag1 && flag2 && flag3)
    {
        d_compartmentDEMIn->colEfficiency[idx4] = d_compartmentDEMIn->colProbability[tix % 16];
    }
    else
        d_compartmentDEMIn->colEfficiency[idx4] = 0.0;
    d_compartmentDEMIn->colFrequency[idx4] = (d_compartmentDEMIn->DEMCollisionData[tix] * timeStep) / demTimeStep;

    d_compartmentOut->aggregationKernel[idx4] = d_aggCompVar->aggKernelConst * d_compartmentDEMIn->colFrequency[idx4] * d_compartmentDEMIn->colEfficiency[idx4];
    // printf("Value of kernel at %d and %d is %f \n", tix, tix, d_compartmentOut->aggregationKernel[idx4]);

    d_compVar->aggregationRate[idx4] = d_compartmentIn->sAggregationCheck[idx3] * d_compartmentIn->ssAggregationCheck[idx3s] * d_compartmentOut->aggregationKernel[idx4] * d_compartmentIn->fAll[idx3] * d_compartmentIn->fAll[idx3s];

    d_aggCompVar->depletionThroughAggregation[idx3] += d_compVar->aggregationRate[idx4];
    d_aggCompVar->depletionThroughAggregation[idx3s] += d_compVar->aggregationRate[idx4];
    d_aggCompVar->depletionOfGasThroughAggregation[idx3] = d_aggCompVar->depletionThroughAggregation[idx3] * d_compartmentOut->gasBins[idx3];
    d_aggCompVar->depletionOfLiquidThroughAggregation[idx3] = d_aggCompVar->depletionThroughAggregation[idx3] * d_compartmentOut->liquidBins[idx3];
    __syncthreads();

    for (int i = 0; i < nFirstSolidBins; i++)
    {
        for(int j = 0; j < nSecondSolidBins; j++)
        {
            int s12 = (idx3 % nFirstSolidBins) * (bdx / nFirstSolidBins) + (idx3 % nFirstSolidBins);
            int ss12 = (idx3s % nFirstSolidBins) * (bdx / nFirstSolidBins) + (idx3s % nFirstSolidBins);
            if (d_compartmentIn->sInd[s12] == (i+1) && d_compartmentIn->ssInd[ss12] == (j+1))
            {
                int a = i * nFirstSolidBins + j;
                d_aggCompVar->birthThroughAggregation[idx3] += d_compVar->aggregationRate[idx4];
                d_aggCompVar->firstSolidBirthThroughAggregation[idx3] += (d_compartmentIn->vs[idx3 % nFirstSolidBins] + d_compartmentIn->vs[idx3 % nFirstSolidBins]) * d_compVar->aggregationRate[idx4];
                d_aggCompVar->secondSolidBirthThroughAggregation[idx3] += (d_compartmentIn->vs[idx3s % nFirstSolidBins] + d_compartmentIn->vs[idx3s % nFirstSolidBins]) * d_compVar->aggregationRate[idx4];
                d_aggCompVar->liquidBirthThroughAggregation[idx3] += (d_compartmentOut->liquidBins[idx3] + d_compartmentOut->liquidBins[idx3s]) * d_compVar->aggregationRate[idx4];
                d_aggCompVar->gasBirthThroughAggregation[idx3] += (d_compartmentOut->gasBins[idx3] + d_compartmentOut->gasBins[idx3s]) * d_compVar->aggregationRate[idx4];
            }
        }
    }

    __syncthreads();

    if (fabs(d_aggCompVar->birthThroughAggregation[idx3]) > 1e-16)
    {
        d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] = d_aggCompVar->firstSolidBirthThroughAggregation[idx3] / d_aggCompVar->birthThroughAggregation[idx3];;
        d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] = d_aggCompVar->secondSolidBirthThroughAggregation[idx3] / d_aggCompVar->birthThroughAggregation[idx3];
    }
    else
    {
        d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] = 0.0;
        d_aggCompVar->secondSolidBirthThroughAggregation[idx3s] = 0.0;
    }

    int val1 = idx3 % nFirstSolidBins; // s
    int val2 = idx3s % nSecondSolidBins; // ss
    // int s3 = val1 * nFirstSolidBins + val2;

    if (val1 == nFirstSolidBins - 1 && val2 == nSecondSolidBins - 1)
    {
        d_aggCompVar->birthAggHighHigh[idx3]  = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1]) / (d_compartmentIn->vs[val1] - d_compartmentIn->vs[val1 -1]);
        d_aggCompVar->birthAggHighHigh[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vss[val2]) / (d_compartmentIn->vs[val2] - d_compartmentIn->vs[val2 -1]);
        d_aggCompVar->birthAggHighHigh[idx3] *= d_aggCompVar->birthThroughAggregation[idx3];

        d_aggCompVar->birthAggHighHighLiq[idx3]  = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1]) / (d_compartmentIn->vs[val1] - d_compartmentIn->vs[val1 -1]);
        d_aggCompVar->birthAggHighHighLiq[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vss[val2]) / (d_compartmentIn->vs[val2] - d_compartmentIn->vs[val2 -1]);
        d_aggCompVar->birthAggHighHighLiq[idx3] *= d_aggCompVar->liquidBirthThroughAggregation[idx3];

        d_aggCompVar->birthAggHighHighGas[idx3]  = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1]) / (d_compartmentIn->vs[val1] - d_compartmentIn->vs[val1 -1]);
        d_aggCompVar->birthAggHighHighGas[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vss[val2]) / (d_compartmentIn->vs[val2] - d_compartmentIn->vs[val2 -1]);
        d_aggCompVar->birthAggHighHighGas[idx3] *= d_aggCompVar->gasBirthThroughAggregation[idx3];
    }

    else if (val2 == nSecondSolidBins - 1)
    {
        d_aggCompVar->birthAggLowHigh[idx3] = (d_compartmentIn->vs[val1 + 1] - d_aggCompVar->firstSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]); 
        d_aggCompVar->birthAggLowHigh[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vss[val2]) / (d_compartmentIn->vs[val2] - d_compartmentIn->vs[val2 -1]);
        d_aggCompVar->birthAggLowHigh[idx3] *= d_aggCompVar->birthThroughAggregation[idx3];

        d_aggCompVar->birthAggLowHighLiq[idx3] = (d_compartmentIn->vs[val1 + 1] - d_aggCompVar->firstSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]); 
        d_aggCompVar->birthAggLowHighLiq[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vss[val2]) / (d_compartmentIn->vs[val2] - d_compartmentIn->vs[val2 -1]);
        d_aggCompVar->birthAggLowHighLiq[idx3] *= d_aggCompVar->liquidBirthThroughAggregation[idx3];

        d_aggCompVar->birthAggLowHighGas[idx3] = (d_compartmentIn->vs[val1 + 1] - d_aggCompVar->firstSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]); 
        d_aggCompVar->birthAggLowHighGas[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vss[val2]) / (d_compartmentIn->vs[val2] - d_compartmentIn->vs[val2 -1]);
        d_aggCompVar->birthAggLowHighGas[idx3] *= d_aggCompVar->gasBirthThroughAggregation[idx3];
    }

    else if (val1 == nFirstSolidBins -1)
    {
        d_aggCompVar->birthAggHighLow[idx3] = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1] ) / (d_compartmentIn->vs[val1] - d_compartmentIn->vs[val1 - 1]);
        d_aggCompVar->birthAggHighLow[idx3] *= (d_compartmentIn->vss[val2 + 1] - d_aggCompVar->secondSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggHighLow[idx3] *= d_aggCompVar->birthThroughAggregation[idx3];

        d_aggCompVar->birthAggHighLowLiq[idx3] = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1] ) / (d_compartmentIn->vs[val1] - d_compartmentIn->vs[val1 - 1]);
        d_aggCompVar->birthAggHighLowLiq[idx3] *= (d_compartmentIn->vss[val2 + 1] - d_aggCompVar->secondSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggHighLowLiq[idx3] *= d_aggCompVar->liquidBirthThroughAggregation[idx3];

        d_aggCompVar->birthAggHighLowGas[idx3] = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1] ) / (d_compartmentIn->vs[val1] - d_compartmentIn->vs[val1 - 1]);
        d_aggCompVar->birthAggHighLowGas[idx3] *= (d_compartmentIn->vss[val2 + 1] - d_aggCompVar->secondSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggHighLowGas[idx3] *= d_aggCompVar->gasBirthThroughAggregation[idx3];

    }

    else
    {
        d_aggCompVar->birthAggLowLow[idx3] = (d_compartmentIn->vs[val1 + 1] - d_aggCompVar->firstSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]);
        d_aggCompVar->birthAggLowLow[idx3] *= (d_compartmentIn->vss[val2 + 1] - d_aggCompVar->secondSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggLowLow[idx3] *= d_aggCompVar->birthThroughAggregation[idx3];

        d_aggCompVar->birthAggHighHigh[idx3]  = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1 + 1] ) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]);
        d_aggCompVar->birthAggHighHigh[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] -d_compartmentIn->vss[val2 + 1]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggHighHigh[idx3] *= d_aggCompVar->birthThroughAggregation[idx3];

        d_aggCompVar->birthAggLowHigh[idx3] = (d_compartmentIn->vs[val1 + 1] - d_aggCompVar->firstSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]); 
        d_aggCompVar->birthAggLowHigh[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vss[val2]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggLowHigh[idx3] *= d_aggCompVar->birthThroughAggregation[idx3];

        d_aggCompVar->birthAggHighLow[idx3] = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1 + 1]) / (d_compartmentIn->vs[val1 +1] - d_compartmentIn->vs[val1]);
        d_aggCompVar->birthAggHighLow[idx3] *= (d_compartmentIn->vss[val2 + 1] - d_aggCompVar->secondSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggHighLow[idx3] *= d_aggCompVar->birthThroughAggregation[idx3];



        d_aggCompVar->birthAggLowLowLiq[idx3] = (d_compartmentIn->vs[val1 + 1] - d_aggCompVar->firstSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]);
        d_aggCompVar->birthAggLowLowLiq[idx3] *= (d_compartmentIn->vss[val2 + 1] - d_aggCompVar->secondSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggLowLowLiq[idx3] *= d_aggCompVar->liquidBirthThroughAggregation[idx3];

        d_aggCompVar->birthAggHighHighLiq[idx3]  = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1 + 1] ) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]);
        d_aggCompVar->birthAggHighHighLiq[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] -d_compartmentIn->vss[val2 + 1]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggHighHighLiq[idx3] *= d_aggCompVar->birthThroughAggregation[idx3];

        d_aggCompVar->birthAggLowHighLiq[idx3] = (d_compartmentIn->vs[val1 + 1] - d_aggCompVar->firstSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]); 
        d_aggCompVar->birthAggLowHighLiq[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vss[val2]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggLowHighLiq[idx3] *= d_aggCompVar->liquidBirthThroughAggregation[idx3];

        d_aggCompVar->birthAggHighLowLiq[idx3] = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1 + 1]) / (d_compartmentIn->vs[val1 +1] - d_compartmentIn->vs[val1]);
        d_aggCompVar->birthAggHighLowLiq[idx3] *= (d_compartmentIn->vss[val2 + 1] - d_aggCompVar->secondSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggHighLowLiq[idx3] *= d_aggCompVar->liquidBirthThroughAggregation[idx3];


        d_aggCompVar->birthAggLowLowGas[idx3] = (d_compartmentIn->vs[val1 + 1] - d_aggCompVar->firstSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]);
        d_aggCompVar->birthAggLowLowGas[idx3] *= (d_compartmentIn->vss[val2 + 1] - d_aggCompVar->secondSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggLowLowGas[idx3] *= d_aggCompVar->gasBirthThroughAggregation[idx3];

        d_aggCompVar->birthAggHighHighGas[idx3]  = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1 + 1]) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]);
        d_aggCompVar->birthAggHighHighGas[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] -d_compartmentIn->vss[val2 + 1]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggHighHighGas[idx3] *= d_aggCompVar->gasBirthThroughAggregation[idx3];

        d_aggCompVar->birthAggLowHighGas[idx3] = (d_compartmentIn->vs[val1 + 1] - d_aggCompVar->firstSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val1 + 1] - d_compartmentIn->vs[val1]); 
        d_aggCompVar->birthAggLowHighGas[idx3] *= (d_aggCompVar->secondSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vss[val2]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggLowHighGas[idx3] *= d_aggCompVar->gasBirthThroughAggregation[idx3];

        d_aggCompVar->birthAggHighLowGas[idx3] = (d_aggCompVar->firstSolidVolumeThroughAggregation[idx3] - d_compartmentIn->vs[val1 + 1]) / (d_compartmentIn->vs[val1 +1] - d_compartmentIn->vs[val1]);
        d_aggCompVar->birthAggHighLowGas[idx3] *= (d_compartmentIn->vss[val2 + 1] - d_aggCompVar->secondSolidVolumeThroughAggregation[idx3]) / (d_compartmentIn->vs[val2 + 1] - d_compartmentIn->vs[val2]);
        d_aggCompVar->birthAggHighLowGas[idx3] *= d_aggCompVar->gasBirthThroughAggregation[idx3];
    }
    
    __syncthreads();

    d_aggCompVar->formationThroughAggregationCA[tlx] = d_aggCompVar->birthAggHighHigh[tlx] + d_aggCompVar->birthAggHighLow[tlx] + d_aggCompVar->birthAggLowHigh[tlx] + d_aggCompVar->birthAggLowLow[tlx];
    d_aggCompVar->formationOfLiquidThroughAggregationCA[tlx] = d_aggCompVar->birthAggHighHighLiq[tlx] + d_aggCompVar->birthAggHighLowLiq[tlx] + d_aggCompVar->birthAggLowHighLiq[tlx] + d_aggCompVar->birthAggLowLowLiq[tlx];
    d_aggCompVar->formationOfGasThroughAggregationCA[tlx] = d_aggCompVar->birthAggHighHighGas[tlx] + d_aggCompVar->birthAggHighLowGas[tlx] + d_aggCompVar->birthAggLowHighGas[tlx] + d_aggCompVar->birthAggLowLowGas[tlx];

}

// ==================== BREAKAGE COMPARTMENT CALCULATIONS ===========================================

__global__ void performBreakageCalculations(PreviousCompartmentIn *d_prevCompIn, CompartmentIn *d_compartmentIn, CompartmentDEMIn *d_compartmentDEMIn, 
                                        CompartmentOut *d_compartmentOut, CompartmentVar *d_compVar, BreakageCompVar *d_brCompVar, 
                                        double time, double timeStep, double initialTime, double demTimeStep, int blx, int tlx, int mbdx, int nFirstSolidBins, int nSecondSolidBins)
{
    int tix = threadIdx.x;
    int bdx = blockDim.x;

    int val1 = tlx % nFirstSolidBins; // s
    int val2 = tix % nSecondSolidBins; // ss
    int s3 = val1 * nFirstSolidBins + val2;

    d_compartmentDEMIn->impactFrequency[tix % nFirstSolidBins] = (d_compartmentDEMIn->DEMImpactData[tix % nFirstSolidBins] * timeStep) / demTimeStep;
    int idx4 = blx * mbdx * bdx + tlx * bdx + tix;

    d_compartmentOut->breakageKernel[idx4] = d_compartmentDEMIn->impactFrequency[tix % nFirstSolidBins] * d_compartmentDEMIn->brProbability[tix % nFirstSolidBins] * d_brCompVar->brkKernelConst;

    d_compVar->breakageRate[idx4] = d_compartmentIn->sCheckB[tlx] * d_compartmentIn->ssCheckB[tix] * d_compartmentOut->breakageKernel[idx4] * d_compartmentIn->fAll[tlx];
    
    __syncthreads();
    
    int idx3 = blx * mbdx + tlx; 
    d_brCompVar->depletionThroughBreakage[idx3] += d_compVar->breakageRate[idx4];
    d_brCompVar->depletionOfLiquidthroughBreakage[idx3] = d_brCompVar->depletionThroughBreakage[idx3] * d_compartmentOut->liquidBins[idx3];
    d_brCompVar->depletionOfGasThroughBreakage[idx3] = d_brCompVar->depletionThroughBreakage[idx3] * d_compartmentOut->gasBins[idx3];

    d_brCompVar->birthThroughBreakage1[idx3] = d_compVar->breakageRate[idx4];

    for (int i = 0; i < nFirstSolidBins; i++)
    {
        for(int j = 0; j < nSecondSolidBins; j++)
        {
            int s12 = (tlx % nFirstSolidBins) * (bdx / nFirstSolidBins) + (tlx % nFirstSolidBins);
            int ss12 = (tix % nFirstSolidBins) * (bdx / nFirstSolidBins) + (tix % nFirstSolidBins);
            if (d_compartmentIn->sIndB[s12] == (i+1) && d_compartmentIn->ssIndB[ss12] == (j+1))
            {
                int a = i * nFirstSolidBins + j;
                d_brCompVar->birthThroughBreakage2[a] += d_compVar->aggregationRate[idx4];
                d_brCompVar->firstSolidBirthThroughBreakage[a] += (d_compartmentIn->vs[tlx % nFirstSolidBins] + d_compartmentIn->vs[tlx % nFirstSolidBins]) * d_compVar->breakageRate[idx4];
                d_brCompVar->secondSolidBirthThroughBreakage[a] += (d_compartmentIn->vs[tix % nFirstSolidBins] + d_compartmentIn->vs[tix % nFirstSolidBins]) * d_compVar->breakageRate[idx4];
                d_brCompVar->liquidBirthThroughBreakage2[a] += (d_compartmentOut->liquidBins[tlx] + d_compartmentOut->liquidBins[tix]) * d_compVar->breakageRate[idx4];
                d_brCompVar->gasBirthThroughBreakage2[a] += (d_compartmentOut->gasBins[tlx] + d_compartmentOut->gasBins[tix]) * d_compVar->breakageRate[idx4];

                if (fabs(d_brCompVar->birthThroughBreakage2[a]) > 1e-16)
                {
                    d_brCompVar->firstSolidVolumeThroughBreakage[a] = d_brCompVar->firstSolidBirthThroughBreakage[a] / d_brCompVar->birthThroughBreakage2[a];
                    d_brCompVar->secondSolidVolumeThroughBreakage[a] = d_brCompVar->secondSolidBirthThroughBreakage[a] / d_brCompVar->birthThroughBreakage2[a];
                }
            }
        }
    }
    __syncthreads();
    d_brCompVar->liquidBirthThroughBreakage1[tix] += (d_compartmentOut->liquidBins[tlx] * (d_compVar->volumeBins[tix] / d_compVar->volumeBins[tlx])) * d_compVar->breakageRate[idx4];
    d_brCompVar->gasBirthThroughBreakage1[tix] += (d_compartmentOut->gasBins[tlx] * (d_compVar->volumeBins[tix] / d_compVar->volumeBins[tlx])) * d_compVar->breakageRate[idx4];

    double value1 = 0.0;
    double value2 = 0.0;

    value1 = fabs(d_compartmentIn->sLow[tix] - d_brCompVar->firstSolidBirthThroughBreakage[tix]);
    value1 = d_compartmentIn->sHigh[tix] - d_compartmentIn->sLow[tix] - value1;
    value1 /= d_compartmentIn->sHigh[tix] - d_compartmentIn->sLow[tix];

    value2 = fabs(d_compartmentIn->ssLow[tix] - d_brCompVar->secondSolidVolumeThroughBreakage[tix]);
    value2 = d_compartmentIn->ssHigh[tix] - d_compartmentIn->ssLow[tix] - value2;
    value2 /= d_compartmentIn->ssHigh[tix] - d_compartmentIn->ssLow[tix];

    d_brCompVar->fractionBreakage00[tix] = value1 / value2;

    value2 = fabs(d_compartmentIn->ssHigh[tix] - d_brCompVar->secondSolidVolumeThroughBreakage[tix]);
    value2 = d_compartmentIn->ssHigh[tix] - d_compartmentIn->ssLow[tix] - value2;
    value2 /= d_compartmentIn->ssHigh[tix] - d_compartmentIn->ssLow[tix];
    
    d_brCompVar->fractionBreakage01[tix] = value1 / value2;

    value1 = fabs(d_compartmentIn->sHigh[tix] - d_brCompVar->firstSolidVolumeThroughBreakage[tix]);
    value1 = d_compartmentIn->sHigh[tix] - d_compartmentIn->sLow[tix] - value1;
    value1 /= d_compartmentIn->sHigh[tix] - d_compartmentIn->sLow[tix];

    d_brCompVar->fractionBreakage11[tix] = value1 / value2;

    value2 = fabs(d_compartmentIn->ssLow[tix] - d_brCompVar->secondSolidVolumeThroughBreakage[tix]);
    value2 = d_compartmentIn->ssHigh[tix] - d_compartmentIn->ssLow[tix] - value2;
    value2 /= d_compartmentIn->ssHigh[tix] - d_compartmentIn->ssLow[tix];

    d_brCompVar->fractionBreakage10[tix] = value1 / value2;

    __syncthreads();

    if (val1 == nFirstSolidBins - 1 && val2 == nSecondSolidBins - 1)
    {
            d_brCompVar->formationThroughBreakageCA[s3] += d_brCompVar->birthThroughBreakage2[s3] * d_brCompVar->fractionBreakage11[s3];
            d_brCompVar->formationOfLiquidThroughBreakageCA[s3] += d_brCompVar->liquidBirthThroughBreakage2[s3] * d_brCompVar->fractionBreakage11[s3];
            d_brCompVar->formationOfGasThroughBreakageCA[s3] += d_brCompVar->gasBirthThroughBreakage2[s3] * d_brCompVar->fractionBreakage11[s3];

    }

    else if (val2 == nSecondSolidBins - 1)
    {
        d_brCompVar->formationThroughBreakageCA[s3] += d_brCompVar->birthThroughBreakage2[s3] * d_brCompVar->fractionBreakage01[s3];
        d_brCompVar->formationOfLiquidThroughBreakageCA[s3] += d_brCompVar->liquidBirthThroughBreakage2[s3] * d_brCompVar->fractionBreakage01[s3];
        d_brCompVar->formationOfGasThroughBreakageCA[s3] += d_brCompVar->gasBirthThroughBreakage2[s3] * d_brCompVar->fractionBreakage01[s3];

    }

    else if (val1 == nFirstSolidBins -1)
    {
        d_brCompVar->formationThroughBreakageCA[s3] += d_brCompVar->birthThroughBreakage2[s3] * d_brCompVar->fractionBreakage10[s3];
        d_brCompVar->formationOfLiquidThroughBreakageCA[s3] += d_brCompVar->liquidBirthThroughBreakage2[s3] * d_brCompVar->fractionBreakage10[s3];
        d_brCompVar->formationOfGasThroughBreakageCA[s3] += d_brCompVar->gasBirthThroughBreakage2[s3] * d_brCompVar->fractionBreakage10[s3];

    }

    else
    {
        d_brCompVar->formationThroughBreakageCA[s3] += d_brCompVar->birthThroughBreakage2[s3] * d_brCompVar->fractionBreakage00[s3];
        d_brCompVar->formationOfLiquidThroughBreakageCA[s3] += d_brCompVar->liquidBirthThroughBreakage2[s3] * d_brCompVar->fractionBreakage00[s3];
        d_brCompVar->formationOfGasThroughBreakageCA[s3] += d_brCompVar->gasBirthThroughBreakage2[s3] * d_brCompVar->fractionBreakage00[s3];
    }
}

// ============ Constructors for the Classes =================


CompartmentVar :: CompartmentVar(unsigned int nX2, unsigned int nX5, unsigned int check)
{
    if (check == 0)
    {
        internalLiquid = alloc_double_vector(nX2);
        externalLiquid = alloc_double_vector(nX2);
        externalLiquidContent = alloc_double_vector(nX2);
        volumeBins = alloc_double_vector(nX2);
        aggregationRate = alloc_double_vector(nX5);
        breakageRate = alloc_double_vector(nX5);
        particleMovement = alloc_double_vector(nX2);
        liquidMovement = alloc_double_vector(nX2);
        gasMovement = alloc_double_vector(nX2);
        liquidBins = alloc_double_vector(nX2);
        gasBins = alloc_double_vector(nX2);
    }

    else if (check == 1)
    {
        internalLiquid = device_alloc_double_vector(nX2);
        externalLiquid = device_alloc_double_vector(nX2);
        externalLiquidContent = device_alloc_double_vector(nX2);
        volumeBins = device_alloc_double_vector(nX2);
        aggregationRate = device_alloc_double_vector(nX5);
        breakageRate = device_alloc_double_vector(nX5);
        particleMovement = device_alloc_double_vector(nX2);
        liquidMovement = device_alloc_double_vector(nX2);
        gasMovement = device_alloc_double_vector(nX2);
        liquidBins = device_alloc_double_vector(nX2);
        gasBins = device_alloc_double_vector(nX2);
    }

    else
        printf("\n Wrong Value of check passed in Compartment Var call \n");
}

CompartmentIn :: CompartmentIn (unsigned int nX2, unsigned int nX5, unsigned int check)
{
    if (check == 0)
    {
        fAll = alloc_double_vector(nX5 / nX2);
        fLiquid = alloc_double_vector(nX5 / nX2);
        fGas = alloc_double_vector(nX5 / nX2);
        liquidAdditionRate = 0.0;
        vs = alloc_double_vector(nX2);
        vss = alloc_double_vector(nX2);
        sMeshXY = alloc_double_vector(nX2);
        ssMeshXY = alloc_double_vector(nX2);
        sAggregationCheck = alloc_integer_vector(nX2);
        ssAggregationCheck = alloc_integer_vector(nX2);;
        sInd = alloc_integer_vector(nX2);;
        ssInd = alloc_integer_vector(nX2);;
        sIndB = alloc_integer_vector(nX2);;
        ssIndB = alloc_integer_vector(nX2);;
        sLow = alloc_double_vector(nX2);
        sHigh = alloc_double_vector(nX2);
        ssLow = alloc_double_vector(nX2);
        ssHigh = alloc_double_vector(nX2);
        sCheckB = alloc_integer_vector(nX2);;
        ssCheckB = alloc_integer_vector(nX2);;
        diameter = alloc_double_vector(nX2);
    }

    else if (check == 1)
    {
        liquidAdditionRate = 0.0;
        fAll = device_alloc_double_vector(nX5 / nX2);
        fLiquid = device_alloc_double_vector(nX5 / nX2);
        fGas = device_alloc_double_vector(nX5 / nX2);
        vs = device_alloc_double_vector(nX2);
        vss = device_alloc_double_vector(nX2);
        sMeshXY = device_alloc_double_vector(nX2);
        ssMeshXY = device_alloc_double_vector(nX2);
        sAggregationCheck = device_alloc_integer_vector(nX2);
        ssAggregationCheck = device_alloc_integer_vector(nX2);;
        sInd = device_alloc_integer_vector(nX2);;
        ssInd = device_alloc_integer_vector(nX2);;
        sIndB = device_alloc_integer_vector(nX2);;
        ssIndB = device_alloc_integer_vector(nX2);;
        sLow = device_alloc_double_vector(nX2);
        sHigh = device_alloc_double_vector(nX2);
        ssLow = device_alloc_double_vector(nX2);
        ssHigh = device_alloc_double_vector(nX2);
        sCheckB = device_alloc_integer_vector(nX2);;
        ssCheckB = device_alloc_integer_vector(nX2);;
        diameter = device_alloc_double_vector(nX2);
    }

    else
        printf("\n Wrong Value of check passed in CompartmentIn  call \n");
}

PreviousCompartmentIn :: PreviousCompartmentIn(unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        fAllPreviousCompartment = alloc_double_vector(nX5 / nX2);
        flPreviousCompartment = alloc_double_vector(nX5 / nX2);
        fgPreviousCompartment = alloc_double_vector(nX5 / nX2);
        fAllComingIn = alloc_double_vector(nX5 / nX2);
        fgComingIn = alloc_double_vector(nX5 / nX2);
    }

    else if (check == 1)
    {
        fAllPreviousCompartment = device_alloc_double_vector(nX5 / nX2);
        flPreviousCompartment = device_alloc_double_vector(nX5 / nX2);
        fgPreviousCompartment = device_alloc_double_vector(nX5 / nX2);
        fAllComingIn = device_alloc_double_vector(nX5 / nX2);
        fgComingIn = device_alloc_double_vector(nX5 / nX2);
    }
    else
        printf("\n Wrong Value of check passed in PreviousCompartmentIn  call \n");

}

CompartmentDEMIn :: CompartmentDEMIn(unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        DEMDiameter = alloc_double_vector(sqrt(nX2));
        DEMCollisionData = alloc_double_vector(nX2);
        DEMImpactData = alloc_double_vector(sqrt(nX2));
        colProbability = alloc_double_vector(sqrt(nX2));
        brProbability = alloc_double_vector(sqrt(nX2));
        colEfficiency = alloc_double_vector(nX4);
        colFrequency = alloc_double_vector(nX4);
        velocityCol = alloc_double_vector(sqrt(nX2));
        impactFrequency = alloc_double_vector(sqrt(nX2));
        uCriticalCol = 0.0;
        ubreak = 0.0;
    }

    else if (check == 1)
    {
        DEMDiameter = device_alloc_double_vector(sqrt(nX2));
        DEMCollisionData = device_alloc_double_vector(nX2);
        DEMImpactData = device_alloc_double_vector(sqrt(nX2));
        colProbability = device_alloc_double_vector(sqrt(nX2));
        brProbability = device_alloc_double_vector(sqrt(nX2));
        colEfficiency = device_alloc_double_vector(nX4);
        colFrequency = device_alloc_double_vector(nX4);
        velocityCol = device_alloc_double_vector(sqrt(nX2));
        impactFrequency = device_alloc_double_vector(sqrt(nX2));
        uCriticalCol = 0.0;
        ubreak = 0.0;
    }

    else 
        printf("\n Wrong Value of check passed in CompartmentDEMIn call \n");

}

CompartmentOut :: CompartmentOut(unsigned int nX2, unsigned int nX5, unsigned int check)
{
    if (check == 0)
    {
        dfAlldt = alloc_double_vector(nX2);
        dfLiquiddt = alloc_double_vector(nX2);
        dfGasdt = alloc_double_vector(nX2);
        liquidBins = alloc_double_vector(nX5 / nX2);
        gasBins = alloc_double_vector(nX5 / nX2);
        internalVolumeBins = alloc_double_vector(nX5 / nX2);
        externalVolumeBins = alloc_double_vector(nX5 / nX2);
        aggregationKernel = alloc_double_vector(nX5);
        breakageKernel = alloc_double_vector(nX5);
        collisionFrequency = alloc_double_vector(nX5);
        formationThroughAggregation = 0.0;
        depletionThroughAggregation = 0.0;
        formationThroughBreakage = 0.0;
        depletionThroughBreakage = 0.0;
    }

    else if (check == 1)
    {
        dfAlldt = device_alloc_double_vector(nX2);
        dfLiquiddt = device_alloc_double_vector(nX2);
        dfGasdt = device_alloc_double_vector(nX2);
        liquidBins = device_alloc_double_vector(nX5 / nX2);
        gasBins = device_alloc_double_vector(nX5 / nX2);
        internalVolumeBins = device_alloc_double_vector(nX5 / nX2);
        externalVolumeBins = device_alloc_double_vector(nX5 / nX2);
        aggregationKernel = device_alloc_double_vector(nX5);
        breakageKernel = device_alloc_double_vector(nX5);
        collisionFrequency = device_alloc_double_vector(nX5);
        formationThroughAggregation = 0.0;
        depletionThroughAggregation = 0.0;
        formationThroughBreakage = 0.0;
        depletionThroughBreakage = 0.0;
    }

    else
        printf("\n Wrong Value of check passed in CompartmentOut call \n");
}

BreakageCompVar :: BreakageCompVar(unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        brkKernelConst = 0.0;
        birthThroughBreakage1 = alloc_double_vector(nX2);
        birthThroughBreakage2 = alloc_double_vector(nX2);
        firstSolidBirthThroughBreakage = alloc_double_vector(nX2);
        secondSolidBirthThroughBreakage = alloc_double_vector(nX2);
        liquidBirthThroughBreakage1 = alloc_double_vector(nX2);
        gasBirthThroughBreakage1 = alloc_double_vector(nX2);
        liquidBirthThroughBreakage2 = alloc_double_vector(nX2);
        gasBirthThroughBreakage2 = alloc_double_vector(nX2);
        firstSolidVolumeThroughBreakage = alloc_double_vector(nX2);
        secondSolidVolumeThroughBreakage = alloc_double_vector(nX2);
        fractionBreakage00 = alloc_double_vector(nX2);
        fractionBreakage01 = alloc_double_vector(nX2);
        fractionBreakage10 = alloc_double_vector(nX2);
        fractionBreakage11 = alloc_double_vector(nX2);
        formationThroughBreakageCA = alloc_double_vector(nX2);
        formationOfLiquidThroughBreakageCA = alloc_double_vector(nX2);
        formationOfGasThroughBreakageCA = alloc_double_vector(nX2);
        transferThroughLiquidAddition = alloc_double_vector(nX2);
        transferThroughConsolidation = alloc_double_vector(nX2);
        depletionThroughBreakage = alloc_double_vector(nX2);
        depletionOfGasThroughBreakage = alloc_double_vector(nX2);
        depletionOfLiquidthroughBreakage = alloc_double_vector(nX2);
    }

    else if (check == 1)
    {
        brkKernelConst = 0.0;
        birthThroughBreakage1 = device_alloc_double_vector(nX2);
        birthThroughBreakage2 = device_alloc_double_vector(nX2);
        firstSolidBirthThroughBreakage = device_alloc_double_vector(nX2);
        secondSolidBirthThroughBreakage = device_alloc_double_vector(nX2);
        liquidBirthThroughBreakage1 = device_alloc_double_vector(nX2);
        gasBirthThroughBreakage1 = device_alloc_double_vector(nX2);
        liquidBirthThroughBreakage2 = device_alloc_double_vector(nX2);
        gasBirthThroughBreakage2 = device_alloc_double_vector(nX2);
        firstSolidVolumeThroughBreakage = device_alloc_double_vector(nX2);
        secondSolidVolumeThroughBreakage = device_alloc_double_vector(nX2);
        fractionBreakage00 = device_alloc_double_vector(nX2);
        fractionBreakage01 = device_alloc_double_vector(nX2);
        fractionBreakage10 = device_alloc_double_vector(nX2);
        fractionBreakage11 = device_alloc_double_vector(nX2);
        formationThroughBreakageCA = device_alloc_double_vector(nX2);
        formationOfLiquidThroughBreakageCA = device_alloc_double_vector(nX2);
        formationOfGasThroughBreakageCA = device_alloc_double_vector(nX2);
        transferThroughLiquidAddition = device_alloc_double_vector(nX2);
        transferThroughConsolidation = device_alloc_double_vector(nX2);
        depletionThroughBreakage = device_alloc_double_vector(nX2);
        depletionOfGasThroughBreakage = device_alloc_double_vector(nX2);
        depletionOfLiquidthroughBreakage = device_alloc_double_vector(nX2);
    }

    else
        printf("\n Wrong Value of check passed in BreakageCompVar call \n");
}

AggregationCompVar :: AggregationCompVar(unsigned int nX2, unsigned int nX4, unsigned int check)
{
    if (check == 0)
    {
        aggKernelConst = 0.0;
        depletionOfGasThroughAggregation = alloc_double_vector(nX2);
        depletionOfLiquidThroughAggregation = alloc_double_vector(nX2);
        birthThroughAggregation = alloc_double_vector(nX2);
        firstSolidBirthThroughAggregation = alloc_double_vector(nX2);
        secondSolidBirthThroughAggregation = alloc_double_vector(nX2);
        liquidBirthThroughAggregation = alloc_double_vector(nX2);
        gasBirthThroughAggregation = alloc_double_vector(nX2);
        firstSolidVolumeThroughAggregation = alloc_double_vector(nX2);
        secondSolidVolumeThroughAggregation = alloc_double_vector(nX2);
        birthAggLowLow = alloc_double_vector(nX2);
        birthAggHighHigh = alloc_double_vector(nX2);
        birthAggLowHigh = alloc_double_vector(nX2);
        birthAggHighLow = alloc_double_vector(nX2);
        birthAggLowLowLiq = alloc_double_vector(nX2);
        birthAggHighHighLiq = alloc_double_vector(nX2);
        birthAggLowHighLiq = alloc_double_vector(nX2);
        birthAggHighLowLiq = alloc_double_vector(nX2);
        birthAggLowLowGas = alloc_double_vector(nX2);
        birthAggHighHighGas = alloc_double_vector(nX2);
        birthAggLowHighGas = alloc_double_vector(nX2);
        birthAggHighLowGas = alloc_double_vector(nX2);
        formationThroughAggregationCA = alloc_double_vector(nX2);
        formationOfLiquidThroughAggregationCA = alloc_double_vector(nX2);
        formationOfGasThroughAggregationCA = alloc_double_vector(nX2);
        depletionThroughAggregation = alloc_double_vector(nX2);
    }

    else if (check == 1)
    {
        aggKernelConst = 0.0;
        depletionOfGasThroughAggregation = device_alloc_double_vector(nX2);
        depletionOfLiquidThroughAggregation = device_alloc_double_vector(nX2);
        birthThroughAggregation = device_alloc_double_vector(nX2);
        firstSolidBirthThroughAggregation = device_alloc_double_vector(nX2);
        secondSolidBirthThroughAggregation = device_alloc_double_vector(nX2);
        liquidBirthThroughAggregation = device_alloc_double_vector(nX2);
        gasBirthThroughAggregation = device_alloc_double_vector(nX2);
        firstSolidVolumeThroughAggregation = device_alloc_double_vector(nX2);
        secondSolidVolumeThroughAggregation = device_alloc_double_vector(nX2);
        birthAggLowLow = device_alloc_double_vector(nX2);
        birthAggHighHigh = device_alloc_double_vector(nX2);
        birthAggLowHigh = device_alloc_double_vector(nX2);
        birthAggHighLow = device_alloc_double_vector(nX2);
        birthAggLowLowLiq = device_alloc_double_vector(nX2);
        birthAggHighHighLiq = device_alloc_double_vector(nX2);
        birthAggLowHighLiq = device_alloc_double_vector(nX2);
        birthAggHighLowLiq = device_alloc_double_vector(nX2);
        birthAggLowLowGas = device_alloc_double_vector(nX2);
        birthAggHighHighGas = device_alloc_double_vector(nX2);
        birthAggLowHighGas = device_alloc_double_vector(nX2);
        birthAggHighLowGas = device_alloc_double_vector(nX2);
        formationThroughAggregationCA = device_alloc_double_vector(nX2);
        formationOfLiquidThroughAggregationCA = device_alloc_double_vector(nX2);
        formationOfGasThroughAggregationCA = device_alloc_double_vector(nX2);
        depletionThroughAggregation = device_alloc_double_vector(nX2);
    }
    else
        printf("\n Wrong Value of check passed in BreakageCompVar call \n");
}