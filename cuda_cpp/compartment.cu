#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>
#include <cmath>

#include "parameterData.h"
#include "utility.cuh"
#include "compartment.cuh"

using namespace std;

__global__ void performAggCalculations(PreviousCompartmentIn *prevCompIn, CompartmentIn *compartmentIn, CompartmentDEMIn *compartmentDEMIn, 
                                        CompartmentOut *compartmentOut, CompartmentVar *d_compVar, AggregationCompVar *d_aggCompVar, 
                                        double time, double timeStep, double initialTime)
{
    
}
